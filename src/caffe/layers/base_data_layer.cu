﻿#include <vector>

#include "caffe/layers/base_data_layer.hpp"

namespace caffe {

template <typename Dtype>
void BasePrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  if (prefetch_current_) {
    prefetch_free_.push(prefetch_current_);
  }
  prefetch_current_ = prefetch_full_.pop("Waiting for data");
  // Reshape to loaded data.
  top[0]->ReshapeLike(prefetch_current_->data_);
  top[0]->set_gpu_data(prefetch_current_->data_.mutable_gpu_data());
  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[1]->ReshapeLike(prefetch_current_->label_);
    top[1]->set_gpu_data(prefetch_current_->label_.mutable_gpu_data());
  }
  //Dtype* d = prefetch_current_->roi_.mutable_cpu_data();
  //DLOG(INFO) << d[0] << " " << d[1] << " " << d[2] << " " << d[3];
  if (this->output_roi_) { //加了有问题
	  // Reshape to loaded labels.
	  top[2]->ReshapeLike(prefetch_current_->roi_);
	  top[2]->set_gpu_data(prefetch_current_->roi_.mutable_gpu_data());
  }
  if (this->output_pts_) { //加了有问题
						   // Reshape to loaded labels.
	  top[3]->ReshapeLike(prefetch_current_->pts_);
	  top[3]->set_gpu_data(prefetch_current_->pts_.mutable_gpu_data());
  }
  //d = top[2]->mutable_cpu_data();
  //DLOG(INFO) << d[0] << " " << d[1] << " " << d[2] << " " << d[3];
  if (this->output_weights_) {
    // Reshape to loaded weights.
    top[4]->ReshapeLike(prefetch_current_->weight_);
    // Copy the weights.
    top[4]->set_gpu_data(prefetch_current_->weight_.mutable_gpu_data());
  }
}

INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);

}  // namespace caffe
